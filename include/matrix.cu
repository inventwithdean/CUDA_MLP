#include "matrix.h"

Matrix::Matrix(int rows, int cols) : rows(rows), cols(cols)
{
    size_t size = rows * cols * sizeof(float);
    hipError_t err = hipMallocManaged(&mat, size);
    if (err != hipSuccess)
    {
        printf("hipMallocManaged Failed: %s\n", hipGetErrorString(err));
    }
    FillRandom();
}

Matrix *Matrix::dot(Matrix *other)
{
    // This multiplied by mat
    size_t out_rows = rows;
    size_t out_cols = other->cols;
    Matrix *out = new Matrix(out_rows, out_cols);
    matmul(this->mat, other->mat, out->mat, rows, cols, out_cols);
    return out;
}

Matrix *Matrix::add(Matrix *other)
{
    // Currently, only supports axis 0 broadcasting!
    // In the form of weights(M, N) + biases (1, N)
    size_t out_rows = rows;
    size_t out_cols = cols;
    Matrix *out = new Matrix(rows, cols);
    mat_add_broadcasted(mat, other->mat, out->mat, out_rows, out_cols);
    return out;
}

void Matrix::print()
{
    printf("Matrix: Shape (%d, %d)\n", rows, cols);
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            int idx = i * cols + j;
            printf("%f ", mat[idx]);
        }
        printf("\n");
    }
    printf("\n");
}

void Matrix::FillRandom()
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            int idx = i * cols + j;
            mat[idx] = (float)rand() / (float)(RAND_MAX / 5);
        }
    }
}

Matrix::~Matrix()
{
    hipFree(mat);
}
